#include "hip/hip_runtime.h"
//
// Created by Zero on 2021/2/14.
//


#include "hip/hip_runtime.h"
#include ""
#include "base_libs/lstd/lstd.h"
#include "base_libs/common.h"
#include <stdio.h>
#include <iostream>
#include "render/samplers/sampler.cpp"
#include "render/samplers/independent.cpp"
#include <hip/hip_runtime.h>
#include <cuda/atomic>
#include "render/lights/shader_include.h"
#include "tests/test_light.h"
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

class Sub1 {
public:
    XPU int fun1() {
        return 0;
    }

    XPU int fun2(int a) {
        return a;
    }
};

class Sub2 {
public:
    XPU int fun1() {
        return 1;
    }

    XPU int fun2(int a) {
        return 2 * a;
    }
};

using luminous::lstd::Variant;

class Base : public Variant<Sub1, Sub2> {
public:
    using Variant::Variant;

    XPU int fun1() {
        return dispatch([](auto &&arg) { return arg.fun1(); });
    }

    XPU int fun2(int a) {
        LUMINOUS_VAR_DISPATCH(fun2, a);
    }
};

class BaseP : public Variant<Sub1 *, Sub2 *> {
public:
    using Variant::Variant;

    XPU int fun1() {
        return dispatch([](auto &&arg) { return arg->fun1(); });
    }

    XPU int fun2(int a) {
        LUMINOUS_VAR_PTR_DISPATCH(fun2, a);
    }
};



XPU void testVariant() {
    using namespace std;

    Sub1 s1 = Sub1();
    Sub2 s2 = Sub2();

//    printf("%d s--\n", s1.fun1());
//    printf("%d s2--\n", s2.fun1());

    Base b(s1);

    Base b2(s2);
    printf("%d b1--  %d s1\n", b.fun1(), s1.fun1());
    printf("%d b2--  %d s2\n", b2.fun1(), s2.fun1());
    printf("%d b1 ++--  %d s1\n", b.fun2(9), s1.fun2(9));
    printf("%d b2 ++--  %d s2\n", b2.fun2(8), s2.fun2(8));

//
//    cout << sizeof(b) << endl;
//    cout << sizeof(s2) << endl;
//
////
//    cout << b.fun1() << endl;
//    cout << b.fun2(9) << endl;
//
//    BaseP bp = &s1;
//
//    BaseP bp2 = &s2;
//
//    cout << bp.fun1() << endl;
//    cout << bp.fun2(9) << endl;
//
//    cout << bp2.fun1() << endl;
//    cout << bp2.fun2(9) << endl;
}


extern "C" {
    __global__ void addKernel(int *c, const int *a, const int *b) {
        int i = threadIdx.x;
        c[i] = a[i] + b[i];
//        testVariant();
        printf("C:%d, B:%d, A: %d\n", c[i], b[i], a[i]);
    }

    __global__ void testKernel(int *c) {
        printf("%d \n", threadIdx.x);
    }

    __global__ void test_tex_sample(hipTextureObject_t handle, float u, float v) {
//        auto val = tex2D<uint8_t>(handle, 0, v);
//        auto val2 = tex2D<uint8_t>(handle, 1, v);
        auto val = tex2D<float>(handle, 0, v);
        auto val2 = tex2D<float>(handle, 1, v);
//        printf("%d,%d,%d,%d\n", (uint32_t)val.x,(uint32_t)val.y,(uint32_t)val.z,(uint32_t)val.w);
//        printf("tex2D[0] :%u, tex2D[1] : %u\n",(uint32_t)val,(uint32_t)val2);
        printf("tex2D[0] :%f, tex2D[1] : %f\n",val,val2);
    }


    __global__ void test_light(luminous::Light*light) {
        using namespace luminous;
//        light.print();
        printf("%f\n", light->get<AreaLight>()->padded);
    }

    __global__ void test_area_light(luminous::AreaLight*light) {
        using namespace luminous;
        //        light.print();
        printf("%f\n", light->padded);
    }

    __global__ void test_AL(luminous::AL * light) {
        using namespace luminous;
        //        light.print();
        printf("%f    %d\n", light->padded, light->_type);
    }
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size) {
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void **) &dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void **) &dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void **) &dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}
